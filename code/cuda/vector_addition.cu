
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* a, int N){
    int i;
    for (i = 0; i < N; ++i){
        // Rand number between 0 - 19
        a[i] = rand() % 20;
    }
}

#define N 20

int main(void) {

    //////////////////////
    // Vector Addition
    /////////////////////

    // host copies of a, b, c
	int *a, *b, *c;
    // device copies of a, b, c
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Alloc space for host copies of a, b, c
	// Setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

    for(int i=0; i< N; i++)
    {
        printf("Input vector a + b: %d + %d = %d\n",
                a[i], b[i], a[i] + b[i]);
    }

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU with N blocks
	add<<<N,1>>>(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for(int i=0; i< N; i++)
    {
        printf("Vector Addition Result : %d \n", c[i]);
    }

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
