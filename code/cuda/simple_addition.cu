
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c){
    *c = *a + *b;
}


int main(void) {

    //////////////////////
    // Integer Addition
    /////////////////////
    int a, b, c;
    int *d_a, *d_b, *d_c;

    int size = sizeof(int);

    // Allocate space for device copies
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a = 10;
    b = 7;

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Launch the kernel
    add<<<1,1>>>(d_a, d_b, d_c);

    // Copy results back
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    printf("Int Addition :  %d + %d = %d \n",a , b, c);

    // Clean up
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
